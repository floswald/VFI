#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////
///
/// @file main.cu
///
/// @brief File containing main function for the VFI problem.
///
/// @author Eric M. Aldrich \n
///         ealdrich@ucsc.edu
///
/// @version 1.0
///
/// @date 23 Oct 2012
///
/// @copyright Copyright Eric M. Aldrich 2012 \n
///            Distributed under the Boost Software License, Version 1.0
///            (See accompanying file LICENSE_1_0.txt or copy at \n
///            http://www.boost.org/LICENSE_1_0.txt)
///
//////////////////////////////////////////////////////////////////////////////

#include "global.h"
#include "auxFuncs.h"
#include "hipblas.h"
#include <iostream>
#include <ctime>
#include <typeinfo>
#include <fstream>

using namespace std;

#include "vfStep.cu"

//////////////////////////////////////////////////////////////////////////////
///
/// @fn main()
///
/// @brief Main function for the VFI problem.
///
/// @details This function solves a standard neoclassical growth model with
/// value function iteration on a GPU.
///
/// @details See Aldrich, Eric M., Jesus Fernandez-Villaverde,
/// A. Ronald Gallant and Juan F. Rubio-Ramirez (2011), "Tapping the
/// supercomputer under your desk: Solving dynamic equilibrium models with
/// graphics processors", Journal of Economic Dynamics & Control, 35, 386-393.
///
/// @returns 0 upon successful completion, 1 otherwise.
///
//////////////////////////////////////////////////////////////////////////////
int main()
{ 

  // Admin
  int imax;
  REAL diff = 1.0;
  REAL negOne = -1.0;

  // Load parameters
  parameters params;
  params.load("../parameters.txt");
  int nk = params.nk;
  int nz = params.nz;

  // Time the GPU startup overhead
  double tic = curr_second();
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  double toc = curr_second();
  double startTime = toc - tic;

  // Pointers to variables in device memory
  REAL *K, *Z, *P, *V0, *V, *G, *Vtemp;

  // Allocate variables in device memory
  tic = curr_second(); // Start the timer for solution
  size_t sizeK = nk*sizeof(REAL);
  size_t sizeZ = nz*sizeof(REAL);
  size_t sizeP = nz*nz*sizeof(REAL);
  size_t sizeV = nk*nz*sizeof(REAL);
  size_t sizeG = nk*nz*sizeof(REAL);
  hipMalloc((void**)&K, sizeK);
  hipMalloc((void**)&Z, sizeZ);
  hipMalloc((void**)&P, sizeP);
  hipMalloc((void**)&V0, sizeV);
  hipMalloc((void**)&Vtemp, sizeV);
  hipMalloc((void**)&V, sizeV);
  hipMalloc((void**)&G, sizeG);

  // Blocking
  const int block_size = 4; ///< Block size for CUDA kernel.
  dim3 dimBlockV(block_size, nz);
  dim3 dimGridV(nk/block_size,1);
 
  // Compute TFP grid, capital grid and initial VF
  REAL hK[nk], hZ[nz], hP[nz*nz], hV0[nk*nz];
  ar1(params, hZ, hP);
  kGrid(params, hZ, hK);
  vfInit(params, hZ, hV0);

  // Copy capital grid, TFP grid and transition matrix to GPU memory
  hipMemcpy(K, hK, sizeK, hipMemcpyHostToDevice);
  hipMemcpy(Z, hZ, sizeZ, hipMemcpyHostToDevice);
  hipMemcpy(P, hP, sizeP, hipMemcpyHostToDevice);
  hipMemcpy(V0, hV0, sizeV, hipMemcpyHostToDevice);

  // Iterate on the value function
  int count = 0;
  while(fabs(diff) > params.tol){
    vfStep<<<dimGridV,dimBlockV>>>(params,K,Z,P,V0,V,G);
    if(typeid(realtype) == typeid(singletype)){
      hipblasSaxpy(handle, nk*nz, (float*)&negOne, (float*)V, 1, (float*)V0, 1);
      hipblasIsamax(handle, nk*nz, (float*)V0, 1, &imax);
    } else if(typeid(realtype) == typeid(doubletype)){
      hipblasDaxpy(handle, nk*nz, (double*)&negOne, (double*)V, 1, (double*)V0, 1);
      hipblasIdamax(handle, nk*nz, (double*)V0, 1, &imax);
    }
    hipMemcpy(&diff, V0+imax, sizeof(REAL), hipMemcpyDeviceToHost);
    Vtemp = V0;
    V0 = V;
    V = Vtemp;
    ++count;
  }
  V = V0;
  
  // Compute solution time
  toc = curr_second();
  double solTime  = toc - tic;

  // Copy value and policy functions to host memory
  REAL* hV = new REAL[nk*nz];
  REAL* hG = new REAL[nk*nz];
  hipMemcpy(hV, V, sizeV, hipMemcpyDeviceToHost);
  hipMemcpy(hG, G, sizeG, hipMemcpyDeviceToHost);

  // Free variables in device memory
  hipFree(K);
  hipFree(Z);
  hipFree(P);
  hipFree(V0);
  hipFree(V);
  hipFree(Vtemp);
  hipFree(G);
  hipblasDestroy(handle);

  // Write to file (row major)
  ofstream fileStartTime, fileSolTime, fileTotalTime, fileValue, filePolicy;
  fileValue.precision(7);
  filePolicy.precision(7);
  fileStartTime.open("startTimeCUDA-C.dat");
  fileSolTime.open("solTimeCUDA-C.dat");
  fileTotalTime.open("totalTimeCUDA-C.dat");
  fileValue.open("valFunCUDA-C.dat");
  filePolicy.open("polFunCUDA-C.dat");
  fileStartTime << startTime << endl;
  fileSolTime << solTime << endl;
  fileSolTime << startTime+solTime << endl;
  fileValue << nk << endl;
  fileValue << nz << endl;
  filePolicy << nk << endl;
  filePolicy << nz << endl;
  for(int jx = 0 ; jx < nz ; ++jx){
    for(int ix = 0 ; ix < nk ; ++ix){
      fileValue << hV[ix*nz+jx] << endl;
      filePolicy << hG[ix*nz+jx] << endl;
    }
  }  
  fileStartTime.close();
  fileSolTime.close();
  fileTotalTime.close();
  fileValue.close();
  filePolicy.close();

  cout << "done after " << solTime << endl;

  return 0;

}
